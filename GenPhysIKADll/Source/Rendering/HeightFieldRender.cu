#include "hip/hip_runtime.h"
#pragma once

#include <GL/glew.h>
#include "HeightFieldRender.h"
#include "Framework/Topology/HeightField.h"
#include "Core/Vector.h"
#include "Core/Utility.h"
#include "Framework/Framework/Node.h"
#include "OpenGLContext.h"
#include "Color.h"


namespace PhysIKA
{
	IMPLEMENT_CLASS(HeightFieldRenderModule)

	HeightFieldRenderModule::HeightFieldRenderModule()
		: VisualModule()
		, m_mode(HeightFieldRenderModule::Instance)
		, m_color(Vector3f(0.8, 0.8, 0.8))
	{
	}

	HeightFieldRenderModule::~HeightFieldRenderModule()
	{
	}

	bool HeightFieldRenderModule::initializeImpl()
	{
		Node* parent = getParent();
		if (parent == NULL)
		{
			Log::sendMessage(Log::Error, "Should insert this module into a node!");
			return false;
		}

		auto pSet = TypeInfo::CastPointerDown<HeightField<DataType3f>>(parent->getTopologyModule());
		if (pSet == nullptr)
		{
			Log::sendMessage(Log::Error, "HeightFieldRenderModule: The topology module is not supported!");
			return false;
		}


		Log::sendMessage(Log::Info, "HeightFieldRenderModule successfully initialized!");
	}

	__global__ void PRM_MappingColor(
		DeviceArray<glm::vec3> color,
		DeviceArray<Vector3f> index,
		float minIndex,
		float maxIndex)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= color.size()) return;

		float index_i = index[tId].norm();

		index_i = index_i > maxIndex ? maxIndex : index_i;
		index_i = index_i < minIndex ? minIndex : index_i;

		float a = (index_i - minIndex) / (maxIndex - minIndex);

		Color hsv;
		hsv.HSVtoRGB(240, 1-a, 1);

		color[tId] = glm::vec3(hsv.r, hsv.g, hsv.b);
	}

	__global__ void PRM_MappingColor(
		DeviceArray<glm::vec3> color,
		DeviceArray<float> index,
		float refV,
		float minIndex,
		float maxIndex)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= color.size()) return;

		float index_i = index[tId];

		index_i = index_i > maxIndex ? maxIndex : index_i;
		index_i = index_i < minIndex ? minIndex : index_i;

		float a = (index_i - refV) / (maxIndex - minIndex);

		Color hsv;
		hsv.HSVtoRGB(a * 120 + 120, 1, 1);

		color[tId] = glm::vec3(hsv.r, hsv.g, hsv.b);
	}

	__global__ void SetupTriangles(
		DeviceArray<float3> vertices,
		DeviceArray<float3> normals,
		DeviceArray<float3> colors,
		DeviceArray2D<float> heights,
		DeviceArray2D<float> terrain,
		float dx,
		float dz,
		float3 origin,
		float3 colorWater,
		float3 colorTerrain)
	{
		int i = threadIdx.x + blockIdx.x * blockDim.x;
		int j = threadIdx.y + blockIdx.y * blockDim.y;

		if (i < heights.Nx() - 1 && j < heights.Ny() - 1)
		{
			int id = i + j * (heights.Nx() - 1);

			float3 v1 = origin + make_float3(i*dx, heights(i, j), j*dz);
			float3 v2 = origin + make_float3((i + 1)*dx, heights(i + 1, j), j*dz);
			float3 v3 = origin + make_float3(i*dx, heights(i, j+1), (j+1)*dz);
			float3 v4 = origin + make_float3((i+1)*dx, heights(i+1, j+1), (j+1)*dz);

 			//float3 v1 = origin + make_float3(i*dx, 0.5f, j*dz);
 			//float3 v2 = origin + make_float3((i + 1)*dx, 0.5f, ((j + 1))*dz);
 			//float3 v3 = origin + make_float3(i*dx, 0.5f, (j + 1)*dz);
 			//float3 v4 = origin + make_float3((i + 1)*dx, 0.5f, (j + 1)*dz);

			vertices[3 * (2 * id) + 0] = v1;
			vertices[3 * (2 * id) + 1] = v2;
			vertices[3 * (2 * id) + 2] = v3;

			float3 triN1 = cross(v2 - v1, v3 - v1);
			triN1 = normalize(triN1);

			normals[3 * (2 * id) + 0] = triN1;
			normals[3 * (2 * id) + 1] = triN1;
			normals[3 * (2 * id) + 2] = triN1;

			colors[3 * (2 * id) + 0] = heights(i, j)> terrain(i, j)? colorWater : colorTerrain;
			colors[3 * (2 * id) + 1] = heights(i + 1, j) > terrain(i + 1, j) ? colorWater : colorTerrain;
			colors[3 * (2 * id) + 2] = heights(i, j + 1) > terrain(i, j + 1) ? colorWater : colorTerrain;


			vertices[3 * (2 * id) + 3] = v3;
			vertices[3 * (2 * id) + 4] = v2;
			vertices[3 * (2 * id) + 5] = v4;

			float3 triN2 = cross(v2 - v3, v4 - v3);
			triN2 = normalize(triN2);

			normals[3 * (2 * id) + 3] = triN2;
			normals[3 * (2 * id) + 4] = triN2;
			normals[3 * (2 * id) + 5] = triN2;

			colors[3 * (2 * id) + 3] = heights(i, j + 1) > terrain(i, j + 1) ? colorWater : colorTerrain;
			colors[3 * (2 * id) + 4] = heights(i + 1, j) > terrain(i + 1, j) ? colorWater : colorTerrain;
			colors[3 * (2 * id) + 5] = heights(i + 1, j + 1) > terrain(i + 1, j + 1) ? colorWater : colorTerrain;
		}

		
	}

	void HeightFieldRenderModule::updateRenderingContext()
	{
		Node* parent = getParent();
		if (parent == NULL)
		{
			Log::sendMessage(Log::Error, "Should insert this module into a node!");
			return;
		}

		auto hf = TypeInfo::CastPointerDown<HeightField<DataType3f>>(parent->getTopologyModule());
		if (hf == nullptr)
		{
			Log::sendMessage(Log::Error, "HeightFieldRenderModule: The topology module is not supported!");
			return;
		}


		auto heights = hf->getHeights();
		auto terrain = hf->getTerrain();
		int numOfTriangles = (heights.Nx() - 1)*(heights.Ny() - 1) * 2;
		//printf("heights nx is %d, ny is %d\n", heights.Nx(), heights.Ny());
		vertices.resize(3 * numOfTriangles);
		normals.resize(3 * numOfTriangles);
		colors.resize(3 * numOfTriangles);
		
		uint3 total_size;
		total_size.x = heights.Nx() - 1;
		total_size.y = heights.Ny() - 1;
		total_size.z = 1;

		auto ori = hf->getOrigin();
		
		cuExecute3D(total_size, SetupTriangles,
			vertices,
			normals,
			colors,
			heights,
			terrain,
			hf->getDx(),
			hf->getDz(),
			make_float3(ori[0], ori[1], ori[2]),
			make_float3(0.0, 0.2, 1),
			make_float3(1.0, 0.0, 0.0));
		char str[200];							
		hipDeviceSynchronize();				
		hipError_t err = hipGetLastError();	
		if (err != hipSuccess)					
		{										
			//sprintf(str, "CUDA error: %d : %s at %s:%d \n", err, hipGetErrorString(err), __FILE__, __LINE__);		
			std::string str = hipGetErrorString(err);
			sprintf("CUDA error:  %s \n", str.c_str());
			throw std::runtime_error(std::string(str));																
		}

		if (m_triangleRender == nullptr)
		{
			m_triangleRender = std::make_shared<TriangleRender>();
		}

		if (m_triangleRender->numberOfTrianlges() != numOfTriangles)
		{
			m_triangleRender->resize(numOfTriangles);
		}
		
		m_triangleRender->setVertexArray(vertices);
		m_triangleRender->setColorArray(colors);
		m_triangleRender->setNormalArray(normals);
	}

	void HeightFieldRenderModule::display()
	{
		glMatrixMode(GL_MODELVIEW_MATRIX);
		glPushMatrix();

		glRotatef(m_rotation.x(), m_rotation.y(), m_rotation.z(), m_rotation.w());
		glTranslatef(m_translation[0], m_translation[1], m_translation[2]);
		glScalef(m_scale[0], m_scale[1], m_scale[2]);

		if (m_triangleRender != nullptr)
			m_triangleRender->display();

		glPopMatrix();
	}

	void HeightFieldRenderModule::setRenderMode(RenderMode mode)
	{
		m_mode = mode;
	}

	void HeightFieldRenderModule::setColor(Vector3f color)
	{
		m_color = color;
	}


	void HeightFieldRenderModule::setReferenceColor(float v)
	{
		m_refV = v;
	}

}